#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "../YarnBall.h"
#include "KittenEngine/includes/modules/SymMat.h"

namespace YarnBall {
	using Kit::hess3;

	__device__ inline vec4 inverseTorque(vec3 f, vec4 b) {
		float f2 = length2(f);
		float s = sqrt(f2) + length(b);
		float D = 1 / (f2 - s * s);
		b *= D;
		return normalize(mat4(
			s - f.x, -f.y, -f.z, 0,
			-f.y, s + f.x, 0, f.z,
			-f.z, 0, s + f.x, -f.y,
			0, f.z, -f.y, s - f.x
		) * b);
	}

#define BLOCK_SIZE (256)
	__global__ void cosseratItr(MetaData* data) {
		const int tid = (int)(blockIdx.x * (BLOCK_SIZE - 1) + threadIdx.x) - 1;
		const int numVerts = data->numVerts;
		if (tid >= numVerts || tid < 0) return;

		const float h = data->h;
		const float damping = data->damping / h;
		const auto verts = data->d_verts;
		const auto dxs = data->d_dx;

		// Linear change
		Vertex v0 = verts[tid];

		// Hessian H
		hess3 H = hess3(1 / (v0.invMass * h * h));
		// vel has been overwritten to contain y - pos
		vec3 dx = dxs[tid];
		vec3 f = 1 / (h * h * v0.invMass) * (v0.vel - dx);

		// Special connections energy
		if (v0.connectionIndex >= 0) {
			constexpr float stiffness = 4e1;
			vec3 p0 = verts[v0.connectionIndex].pos;
			vec3 p0dx = dxs[v0.connectionIndex];
			f -= stiffness * ((v0.pos - p0) + (dx - p0dx) + damping * dx);
			H.diag += stiffness * (1 + damping);
		}

		// We need to store absolute position and position updates seperatly for floating point precision
		// If we added these together, the update could be small enough to be rounded out, causing stability issues
		vec3 p1, p1dx;
		float stepLimit = INFINITY;
		vec3 f2(0);
		hess3 H2(0);

		if (v0.flags & (uint32_t)VertexFlags::hasNext) {
			p1 = verts[tid + 1].pos;
			p1dx = dxs[tid + 1];
			stepLimit = data->d_maxStepSize[tid];

			// Cosserat stretching energy
			{
				float invl = 1 / v0.lRest;
				vec3 c = ((p1 - v0.pos) + (p1dx - dx)) * invl - v0.q * vec3(1, 0, 0);

				float k = v0.kStretch * invl;
				float d = k * invl;
				f += k * c - (damping * d) * dx;
				f2 += -k * c - (damping * d) * p1dx;
				d *= 1 + damping;
				H.diag += d;
				H2.diag += d;
			}

			const float fricK = data->kFriction;
			const float invb = 1 / data->barrierThickness;
			const float radius = 2 * data->radius;
			const float fricMu = data->frictionCoeff;
			const auto collisions = data->d_collisions;
			const float kCol = data->kCollision * invb;

			// Collision energy of this segment
			const int numCols = data->d_numCols[tid];
			const auto lastPos = data->d_lastPos;
			for (int i = 0; i < numCols; i++) {
				int colID = collisions[tid + i * numVerts];

				vec3 b0 = lastPos[colID];
				vec3 b1 = lastPos[colID + 1];
				vec3 db0 = dxs[colID];
				vec3 db1 = dxs[colID + 1];

				// Compute collision UV and normal
				vec2 uv = Kit::segmentClosestPoints(
					vec3(0), (p1 - v0.pos) + (p1dx - dx),
					(b0 - v0.pos) + (db0 - dx), (b1 - v0.pos) + (db1 - dx));
				if (!glm::isfinite(uv.x) || !glm::isfinite(uv.y))
					uv = vec2(0.5);

				vec3 dpos = mix(v0.pos, p1, uv.x) - mix(b0, b1, uv.y);
				vec3 ddpos = mix(dx, p1dx, uv.x) - mix(db0, db1, uv.y);
				vec3 normal = dpos + ddpos;
				float d = length(normal);
				normal /= d;

				uv.y = uv.x;
				uv.x = 1 - uv.x;

				// Compute penetration
				d = d - radius;
				d *= invb;
				if (d > 1) continue;	// Not touching
				d = max(d, 1e-3f);		// Clamp to some small value. This is a ratio of the barrier thickness.

				// IPC barrier energy
				float invd = 1 / d;
				float logd = log(d);

				float dH = (-3 + (2 + invd) * invd - 2 * logd) * kCol * invb;
				float ff = -(1 - d) * (d - 1 + 2 * d * logd) * invd * kCol;
				f += (ff * uv.x - damping * dH * uv.x * uv.x * dot(normal, dx)) * normal;
				f2 += (ff * uv.y - damping * dH * uv.y * uv.y * dot(normal, p1dx)) * normal;

				dH *= 1 + damping;
				hess3 op = hess3::outer(normal);
				H += op * (dH * uv.x * uv.x);
				H2 += op * (dH * uv.y * uv.y);

				// Friction
				vec3 u = ddpos - dot(normal, ddpos) * normal;
				float ul = length(u);
				if (ul > 0) {
					float f1 = glm::min(fricK, fricMu * ff / ul);

					op.diag -= 1;

					f -= f1 * uv.x * u;
					H -= op * (Kit::pow2(uv.x) * f1);

					f2 -= f1 * uv.y * u;
					H2 -= op * (Kit::pow2(uv.y) * f1);
				}
			}
		}

		__shared__ vec4 forces[BLOCK_SIZE];
		__shared__ hess3 hessians[BLOCK_SIZE];
		forces[threadIdx.x] = vec4(f2, stepLimit);
		hessians[threadIdx.x] = H2;

		__syncthreads();

		// No reason to keep thread 0 going anymore
		if (!threadIdx.x) return;

		if (v0.flags & (uint32_t)VertexFlags::hasPrev) {
			vec4 v = forces[threadIdx.x - 1];
			stepLimit = min(stepLimit, v.w);
			f += vec3(v);
			H += hessians[threadIdx.x - 1];
		}

		if (v0.invMass != 0) {
			// Local solve
			vec3 delta = data->accelerationRatio * (inverse((mat3)H) * f);
			dx += delta;

			float l = length(dx);
			if (l > stepLimit && l > 0) dx *= stepLimit / l;

			// Apply update
			dxs[tid] = dx;
		}
	}

	__global__ void quaternionItr(MetaData* data) {
		const int tid = threadIdx.x + blockIdx.x * blockDim.x;
		const int numVerts = data->numVerts;
		if (tid >= numVerts || tid < 0) return;

		const auto verts = data->d_verts;
		const auto dxs = data->d_dx;

		// Linear change
		Vertex v0 = verts[tid];

		// Update segment orientation
		// This is done assuming some very very large invMoment (i.e. no inertia so static equilibrium)
		if (!(bool)(v0.flags & (uint32_t)VertexFlags::fixOrientation) != 0 && (v0.flags & (uint32_t)VertexFlags::hasNext)) {
			vec3 dx = dxs[tid];
			vec3 p1 = verts[tid + 1].pos;
			vec3 p1dx = dxs[tid + 1];

			// All this is from an alternate derivation from forced-base hair interpolation.
			v0.pos = ((p1 - v0.pos) + (p1dx - dx)) / v0.lRest;
			v0.pos *= -2 * v0.kStretch;

			vec4 b(0);
			if (v0.flags & (uint32_t)VertexFlags::hasPrev) {
				auto qRest = Kit::Rotor(verts[tid - 1].qRest);
				auto qq = verts[tid - 1].q;
				float s = dot((qq.inverse() * v0.q).v, qRest.v) > 0 ? 1 : -1;
				b -= (verts[tid - 1].kBend * s) * (qq * qRest).v;
			}

			if (v0.flags & (uint32_t)VertexFlags::hasNextOrientation) {
				auto qq = verts[tid + 1].q;
				float s = dot((v0.q.inverse() * qq).v, v0.qRest) > 0 ? 1 : -1;
				b -= (v0.kBend * s) * (verts[tid + 1].q * Kit::Rotor(v0.qRest).inverse()).v;
			}

			v0.q = inverseTorque(v0.pos, b);
			verts[tid].q = v0.q;
		}
	}

	void Sim::iterateCosserat() {
		cosseratItr << <(meta.numVerts + BLOCK_SIZE - 2) / (BLOCK_SIZE - 1), BLOCK_SIZE, 0, stream >> > (d_meta);
		quaternionItr << <(meta.numVerts + 255) / 256, 256, 0, stream >> > (d_meta);
	}
}