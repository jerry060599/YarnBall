#include "hip/hip_runtime.h"
#include "YarnBall.h"
#include <hip/hip_runtime.h>

#include <hip/hip_runtime.h>
#include <>

namespace YarnBall {
	Sim::Sim(int numVerts) {
		if (numVerts < 3) throw std::runtime_error("Too little vertices");

		meta.numVerts = numVerts;
		meta.gravity = vec3(0, -9.8, 0);
		meta.h = maxH;
		meta.drag = 0.2;
		meta.damping = 1e-6;

		meta.radius = 1e-4;
		meta.barrierThickness = 8e-4;
		meta.accelerationRatio = 1;

		meta.kCollision = 1e-5;
		meta.detectionScaler = 2.f;
		meta.frictionCoeff = 0.1f;
		meta.time = 0.f;
		meta.detectionPeriod = 1;
		meta.bvhRebuildPeriod = 8;
		meta.numItr = 8;

		// Initialize vertices
		verts = new Vertex[numVerts];
		for (size_t i = 0; i < numVerts; i++) {
			verts[i].invMass = verts[i].lRest = 1;
			verts[i].vel = vec3(0);
			verts[i].kBend = 5.f;
			verts[i].kStretch = 100.f;
			verts[i].connectionIndex = -1;
			verts[i].flags = (uint32_t)VertexFlags::hasNext;
		}
		verts[numVerts - 1].flags = 0;
	}

	Sim::~Sim() {
		delete[] verts;
		if (stream) {
			hipStreamSynchronize(stream);
			hipStreamDestroy(stream);
		}

		if (vertBuffer) delete vertBuffer;
		if (d_meta) {
			hipFree(meta.d_dx);
			hipFree(meta.d_lastVels);
			hipFree(meta.d_lastSegments);
			hipFree(meta.d_numCols);
			hipFree(meta.d_collisions);
			hipFree(meta.d_bounds);
			hipFree(meta.d_boundColList);
			hipFree(d_meta);
		}
		if (d_error) hipFree(d_error);
		if (stepGraph) hipGraphExecDestroy(stepGraph);
		if (cylMesh) delete cylMesh;
		if (cylMeshHiRes) delete cylMeshHiRes;
	}

	void Sim::configure(float density) {
		const int numVerts = meta.numVerts;

		meta.maxSegLen = 0;
		meta.minSegLen = FLT_MAX;

		// Init mass and orientation
		for (int i = 0; i < numVerts; i++) {
			auto& v = verts[i];

			// Fix flags
			if (i < numVerts - 1) {
				bool hasPrev = v.flags & (uint32_t)VertexFlags::hasNext;
				verts[i + 1].flags = (verts[i + 1].flags & ~(uint32_t)VertexFlags::hasPrev) | (hasPrev ? (uint32_t)VertexFlags::hasPrev : 0);

				// If the segment doesnt exist, then we fix the rotation
				if (!hasPrev) v.flags |= (uint32_t)VertexFlags::fixOrientation;
			}

			if (!(bool)(v.flags & (uint32_t)VertexFlags::hasPrev) && !(bool)(verts[i + 1].flags & (uint32_t)VertexFlags::hasNext))
				throw std::runtime_error("Dangling segment. Yarns must be atleast 2 segments long");

			v.lRest = 1.f / numVerts;
			v.q = Kit::Rotor::identity();
			v.qRest = vec4(0, 0, 0, 1);

			float mass = 0;
			if (v.flags & (uint32_t)VertexFlags::hasPrev)
				mass += verts[i - 1].lRest;

			if (v.flags & (uint32_t)VertexFlags::hasNext) {
				auto& v1 = verts[i + 1];
				vec3 seg0 = v1.pos - v.pos;
				v.lRest = length(seg0);
				if (v.lRest == 0 || !glm::isfinite(v.lRest))
					throw std::runtime_error("0 length segment");
				v.q = Kit::Rotor::fromTo(vec3(1, 0, 0), normalize(seg0));

				mass += v.lRest;

				meta.maxSegLen = max(meta.maxSegLen, v.lRest);
				meta.minSegLen = min(meta.minSegLen, v.lRest);
			}

			mass *= 0.5f * density;

			if (mass != 0)
				v.invMass *= 1 / mass;
			else
				v.invMass = 0;
		}

		// Init rest orientation
		for (int i = 0; i < numVerts - 1; i++) {
			auto& v0 = verts[i];
			auto& v1 = verts[i + 1];
			verts[i].qRest = (vec4)(v0.q.inverse() * v1.q);
		}

		// Mesh for rendering
		cylMesh = Kit::genCylMesh(6, 1, false);
		cylMeshHiRes = Kit::genCylMesh(8, 6, false);

		// Init meta
		hipMalloc(&d_meta, sizeof(MetaData));

		hipMalloc(&d_error, 2 * sizeof(int));
		hipMemset(d_error, 0, 2 * sizeof(int));

		hipMalloc(&meta.d_dx, sizeof(vec3) * numVerts);

		hipMalloc(&meta.d_lastVels, sizeof(vec3) * numVerts);
		hipMemset(meta.d_lastVels, 0, sizeof(vec3) * numVerts);
		hipMalloc(&meta.d_lastSegments, sizeof(Segment) * numVerts);

		hipMalloc(&meta.d_numCols, sizeof(int) * numVerts);
		hipMemset(meta.d_numCols, 0, sizeof(int) * meta.numVerts);
		hipMalloc(&meta.d_collisions, sizeof(Collision) * numVerts * MAX_COLLISIONS_PER_SEGMENT);
		hipMalloc(&meta.d_bounds, sizeof(Kit::LBVH::aabb) * numVerts);
		hipMalloc(&meta.d_boundColList, sizeof(int) * numVerts * MAX_COLLISIONS_PER_SEGMENT);

		vertBuffer = new Kitten::CudaComputeBuffer(sizeof(Vertex), numVerts);
		meta.d_verts = (Vertex*)vertBuffer->cudaPtr;

		hipDeviceSynchronize();
		hipStreamCreate(&stream);
		uploadMeta();
		upload();
		checkCudaErrors(hipGetLastError());
	}

	void Sim::setKStretch(float kStretch) {
		if (!d_meta) throw std::runtime_error("No rest length. Must call configure()");

		// Multiplied by rest length to make energy density consistent.
		// Each segment has l * E energy, where E = C.k.C
		// The l is moved into the kStretch
		for (int i = 0; i < meta.numVerts; i++)
			verts[i].kStretch = kStretch * verts[i].lRest;
	}

	void Sim::setKBend(float kBend) {
		if (!d_meta) throw std::runtime_error("No rest length. Must call configure()");

		// Scaled by the 4 below
		kBend *= 4;

		// Divded by rest length to make energy density consistent.
		// Each segment has l * E energy, where E = C.k.C
		// The l is moved into the kBend, but we also cheated because the darboux vectors
		// in C should have been scaled by 2/l. So in total we end up dividing once.
		for (int i = 0; i < meta.numVerts; i++)
			verts[i].kBend = kBend / verts[i].lRest;
	}

	void Sim::uploadMeta() {
		meta.detectionRadius = meta.detectionScaler * (meta.radius + 0.5f * meta.barrierThickness);

		if (meta.minSegLen < 2 * meta.radius + meta.barrierThickness)
			throw std::runtime_error("Use thinner yarn or use longer segments. (Min seg length must be at least 2 * radius + barrierThickness");

		hipMemcpyAsync(d_meta, &meta, sizeof(MetaData), hipMemcpyHostToDevice, stream);
	}

	void Sim::upload() {
		hipMemcpyAsync(meta.d_verts, verts, sizeof(Vertex) * meta.numVerts, hipMemcpyHostToDevice, stream);
		hipStreamSynchronize(stream);
	}

	void Sim::download() {
		hipMemcpyAsync(verts, meta.d_verts, sizeof(Vertex) * meta.numVerts, hipMemcpyDeviceToHost, stream);
		hipStreamSynchronize(stream);
	}

	__global__ void zeroVels(Vertex* verts, vec3* lastVels, int numVerts) {
		const int tid = threadIdx.x + blockIdx.x * blockDim.x;
		if (tid >= numVerts) return;

		verts[tid].vel = vec3(0);
		lastVels[tid] = vec3(0);
	}

	void Sim::zeroVelocities() {
		zeroVels << <(meta.numVerts + 1023) / 1024, 1024, 0, stream >> > (meta.d_verts, meta.d_lastVels, meta.numVerts);
		checkCudaErrors(hipGetLastError());
	}

	void Sim::checkErrors() {
		checkCudaErrors(hipGetLastError());

		int error[2];
		hipMemcpyAsync(error, d_error, 2 * sizeof(int), hipMemcpyDeviceToHost, stream);
		hipStreamSynchronize(stream);

		if (error[0] == ERROR_MAX_COLLISIONS_PER_SEGMENT_EXCEEDED) {
			if (printErrors) fprintf(stderr, "ERROR: MAX_COLLISIONS_PER_SEGMENT exceeded. Current simulation state may be corrupted!\n");
			throw std::runtime_error("MAX_COLLISIONS_PER_SEGMENT exceeded");
		}
		else if (error[0] != ERROR_NONE) {
			if (printErrors) fprintf(stderr, "ERROR: Undescript error %d\n", error[0]);
			throw std::runtime_error("Indescript error");
		}

		if (printErrors)
			if (error[1] == WARNING_SEGMENT_STRETCH_EXCEEDS_DETECTION_SCALER)
				fprintf(stderr, "WARNING: Excessive segment stretching detected. Missed collisions possible due to insufficient detection radius.\n");
			else if (error[1] == WARNING_SEGMENT_INTERPENETRATION)
				fprintf(stderr, "WARNING: Interpenetration detection. This can be due to unstable contacts\n");
			else if (error[1] != ERROR_NONE)
				fprintf(stderr, "WARNING: Indescript warning %d\n", error[1]);

		if (error[0] != ERROR_NONE) lastErrorCode = error[0];
		if (error[1] != ERROR_NONE) lastWarningCode = error[1];

		// Reset errors
		if (error[0] != 0 || error[1] != 0)
			hipMemsetAsync(d_error, 0, 2 * sizeof(int), stream);
	}
}